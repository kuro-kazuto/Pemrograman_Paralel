//Add Vector Using GPU

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n){
	for (int i = 0; i < n; i++){
		out[i] = a[i] + b[i];
	}
}

int main(){
	float *a, *b, *out;
	float *d_a, *d_b, *d_out;

	//Alokasi Host Memori
	a = (float*)malloc(sizeof(float) * N);
	b = (float*)malloc(sizeof(float) * N);
	out = (float*)malloc(sizeof(float) * N);

	//Inisialisasi Array
	for (int i = 0; i < N; i++){
		a[i] = 1.0f;
		b[i] = 2.0f;
	}

	//Alokasi Device memori
	hipMalloc((void**)&d_a, sizeof(float) * N);
	hipMalloc((void**)&d_b, sizeof(float) * N);
	hipMalloc((void**)&d_out, sizeof(float) * N);

	//Transfer Data dari Host memori ke Device memori
	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

	//Eksekusi Kernel
	vector_add<<<1,1>>>(d_out, d_a, d_b, N);

	//Transfer Data kembali ke Host Memori
	hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

	//Verification
	for (int i = 0; i < N; i++){
		assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
	}
	printf("out[0] = %f\n", out[0]);
	printf("PASSED\n");

	//Dealokasi Device Memori
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);

	//Dealokasi Host Memori
	free(a);
	free(b);
	free(out);

	return 0;
}